#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cstdlib>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include "mpi.h"

#define NODES      2

#ifdef ENABLE_MULTI_MSG
#define MAX_COMM   3000
#else
#define MAX_COMM   1
#endif

#ifdef ENABLE_HEAVY
#define MSG_SIZE   5500000 //! 22MB
#else
#define MSG_SIZE   100     //! 25xINT
#endif

__global__ void verifyGPUBuffers(int *send_buffer,
                                 int *reduce) {
  for (int i = 0; i < MSG_SIZE; i++) {
    if (send_buffer[i] != i) {
      printf("%d is failed to verified; %d\n", i, send_buffer[i]);
      *reduce += 1;
      break;
    }
  }

  if (*reduce > 0) {
    printf("verifying failed\n");
  }
}

void initializeBuffers(int *send_buffer) {
  for (int i = 0; i < MSG_SIZE; i++) {
    send_buffer[i] = i;
  }
}

void printBuffer(int *buffer) {
  for (int i = 0; i < MSG_SIZE; i++) {
    printf("\tbuffer[%d] = %d ", i, buffer[i]);
  }
  printf("\n");
}

void verifyRecvedBuffers(int *send_buffer,
                         int *reduce) {
  for (int i = 0; i < MSG_SIZE; i++) {
    if (send_buffer[i] != i) {
      printf("%d is failed to verified; %d\n", i, send_buffer[i]);
      *reduce += 1;
      break;
    }
  }

  if (*reduce > 0) {
    printf("verifying failed\n");
  }
}

int main(int argc, char** argv) {
  int rank, p;
  int *buffer;
  int supportProvided;

  MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &supportProvided);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &p);

  printf("Welcome to MPI world. %d out of %d processors\n",
         rank, p);
  printf("Number of nodes: %d, Number of msgs: %d,"
         "Msg size: %d\n", NODES, MAX_COMM, MSG_SIZE);
#ifdef PARALLEL_MSG_MODE
  omp_set_num_threads(56);
  printf("Parallel mode is enabled. Used number of threads is 56\n");
#endif

  //! Initialize buffers.
  buffer = (int *) malloc(MSG_SIZE*sizeof(int));

  if (rank == 0) { ///< Rank0 node.
    printf("RANK 0: Initialize msg..\n");
    initializeBuffers(buffer);
    printf("RANK 0: Send msg..\n");
#ifdef PRINT_BUFFER
    printf("RANK 0: Printing send-msg %d-th\n", i);
    printBuffer(buffer);
#endif
    //! Send the msg one by one.
    for (int neigh = 1; neigh < NODES; neigh++) {
#ifdef PARALLEL_MSG_MODE
      #pragma omp parallel for
#endif
      for (int i = 0; i < MAX_COMM; i++) {
        printf("RANK 0: Sending msg %d-th\n", i);
        MPI_Send(buffer, MSG_SIZE, MPI_INT,
                 neigh, 0, MPI_COMM_WORLD);
        printf("RANK 0: Sending msg %d-th to %d: done\n", i, neigh);
      }
    }
    printf("RANK 0: Rank 0 is done\n");
  } else { ///< Not rank0 nodes.
    //! Initialize receiver-side buffers.
    memset(buffer, 0, sizeof(int)*MSG_SIZE);
#ifdef PRINT_BUFFER
    printf("RANK %d: Print recv buffer before recving\n", rank);
    printBuffer(buffer);
#endif
    int reduce;
    //! Receive the msg one by one.
    for (int i = 0; i < MAX_COMM; i++) {
      reduce = 0;
      printf("RANK %d: Tries to recv %d-th msg (size: %d)\n", rank, i, MSG_SIZE);
      MPI_Recv(buffer, MSG_SIZE, MPI_INT, 0,
               0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
#ifdef PRINT_BUFFER
      printf("RANK %d: Print recv %d-th msg\n", rank, i);
      printBuffer(buffer);
#endif
      printf("Starts to verifying.. %d-th msg\n", i);
      verifyRecvedBuffers(buffer, &reduce); 
      printf("Verified done.. %d-th msg\n", i);
    }
    printf("RANK %d: Received msg\n", rank);

    //! Copy to GPU.
    int* gpu_buffer;
    hipMalloc((void **)&gpu_buffer, sizeof(int)*MSG_SIZE);
    hipMemcpy(gpu_buffer, buffer, sizeof(int)*MSG_SIZE, hipMemcpyHostToDevice);
    printf("RANK %d: Verify the copied data from cpu to gpu..\n", rank);
    reduce = 0;
    verifyGPUBuffers<<<1,1>>>(gpu_buffer, &reduce);
    printf("RANK %d: All jobs are done\n", rank);
    hipDeviceSynchronize();
  }

  MPI_Finalize();
  free(buffer);
  return 0;
}
